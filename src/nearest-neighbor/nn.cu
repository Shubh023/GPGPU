#include "hip/hip_runtime.h"
#include <iostream>

#include "nn.hh"

#define cudaCheckError() {                                                   \
    hipError_t e=hipGetLastError();                                        \
    if(e!=hipSuccess) {                                                     \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__ , __LINE__,            \
               hipGetErrorString(e));                                       \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
}

namespace irgpu {

__global__ void nearest_centroid(double *descriptor, double *centroid, int *pred) {
    return;
}

std::vector<int>
assign_centroids(const std::vector<histogram_t>& h_descriptors, 
                 const std::vector<histogram_t>& h_centroids) {

    double *d_descriptors;
    int l_desc = h_descriptors.size() * DESC_SIZE;
    hipMalloc(&d_descriptors, l_desc * sizeof(double)); 
    cudaCheckError();

    double *d_centroids;
    int l_cent = h_centroids.size() * DESC_SIZE;
    hipMalloc(&d_centroids, l_cent * sizeof(double)); 
    cudaCheckError();

    auto h_assignments = std::vector<int>(h_descriptors.size());
    int *d_assignments;
    hipMalloc(&d_assignments, h_assignments.size() * sizeof(double)); 
    cudaCheckError();

    hipMemcpy(d_descriptors, &h_descriptors[0], l_desc * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, &h_centroids[0], l_cent * sizeof(double),
               hipMemcpyHostToDevice);
    cudaCheckError();

    hipFree(d_descriptors);
    hipFree(d_centroids);
    cudaCheckError();

    return h_assignments;
}

}