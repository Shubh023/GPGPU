#include "nn.hh"

#define cudaCheckError() {                                                   \
    hipError_t e=hipGetLastError();                                        \
    if(e!=hipSuccess) {                                                     \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__ , __LINE__,            \
               hipGetErrorString(e));                                       \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
}

namespace irgpu {

std::vector<int>
assign_centroids(const std::vector<histogram_t>& h_descriptors, 
                 const std::vector<histogram_t>& h_centroids) {

    double *d_descriptors;
    double *d_centroids;
    int l_desc = h_descriptors.size() * 256;
    int l_cent = h_centroids.size() * 256;
    
    hipMalloc(&d_descriptors, l_desc * sizeof(double)); 
    hipMalloc(&d_centroids, l_cent * sizeof(double)); 
    cudaCheckError();

    hipMemcpy(d_descriptors, &h_descriptors[0], l_desc * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, &h_centroids[0], l_cent * sizeof(double),
               hipMemcpyHostToDevice);
    cudaCheckError();

    hipFree(d_descriptors);
    hipFree(d_centroids);
    cudaCheckError();

    return std::vector<int>();
}

}