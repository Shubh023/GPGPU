#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include "nn2.hh"

#define cudaCheckError() {                                                   \
    hipError_t e=hipGetLastError();                                        \
    if(e!=hipSuccess) {                                                     \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__ , __LINE__,            \
               hipGetErrorString(e));                                       \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
}

namespace irgpu {


__global__ void l2_sq(double *mat1, double *mat2, double *l2_sq,
                      int M, int N, int P) {

    int tile_width = blockDim.x;  // square tile

    // Need to split the shared buffer to use two.
    extern __shared__ double tiles[];
    double *tile1 = (double*) tiles;
    double *tile2 = (double*) &tiles[tile_width * tile_width];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Indexes in the grid
    int row = by*blockDim.y + ty;
    int col = bx*blockDim.x + tx;

    double p_sum = 0;     
    for (int k = 0; k < ceil(N / (float) tile_width); k++) {

        // Load into shared memory and check boundaries
        if (row < M && (tx + k*tile_width) < N) {
            tile1[ty*blockDim.y + tx] = mat1[row*N + tx + k*tile_width];    // [row][tx + patch_shift]
        }
        if ((ty + k * tile_width) < N && col < P) {
            tile2[ty*blockDim.y + tx] = mat2[(ty + k*tile_width)*P + col];  // [ty + patch_shift][col]
        }
        //printf("%d %d %f %f %f %f\n", p1, p2, tile1[ty*blockDim.y + tx], tile2[ty*blockDim.y + tx],
        //    mat1[row*N + tx + k*tile_width],mat2[(ty + k*tile_width)*P + col]);
        __syncthreads();

        for (int l = 0; l < tile_width; l++) {
            double diff = tile1[ty*blockDim.y + l] - tile2[l*blockDim.y + tx];
            p_sum += diff*diff;
        }
        __syncthreads();
    }

    if (row < M && col < P) { 
        l2_sq[row*P + col] = p_sum;
    }
}

std::vector<int>
assign_centroids2(const std::vector<histogram_t>& h_descriptors, 
                  const std::vector<double>& h_centroids) {

    std::cout << "Lancer\n";
    std::cout << h_centroids.size() << "\n";
    int n_desc = h_descriptors.size();
    double *d_descriptors;
    hipMalloc(&d_descriptors, n_desc * DESC_DIM * sizeof(double)); 
    cudaCheckError();

    int n_cent = h_centroids.size() / 256;
    double *d_centroids;
    hipMalloc(&d_centroids, n_cent * DESC_DIM * sizeof(double)); 
    cudaCheckError();

    auto h_l2_squared = std::vector<double>(n_desc * n_cent);
    double *d_l2_squared; 
    hipMalloc(&d_l2_squared, n_desc * n_cent * sizeof(double)); 
    cudaCheckError();

    auto h_assignments = std::vector<int>(n_desc);
    int *d_assignments;
    hipMalloc(&d_assignments, n_desc * sizeof(int)); 
    cudaCheckError();

    hipMemcpy(d_descriptors, &h_descriptors[0], n_desc * DESC_DIM * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, &h_centroids[0], n_cent * DESC_DIM * sizeof(double),
               hipMemcpyHostToDevice);
    cudaCheckError();

    // need to adjust because of shared memory usage -> SM
    //dim3 block_dim(32, 32);
    dim3 block_dim(32, 32);
    dim3 grid_dim((n_cent + block_dim.x - 1) / block_dim.x,
                  (n_desc + block_dim.y - 1) / block_dim.y);
    int patch_mem = block_dim.x*block_dim.y*sizeof(double);
    std::cout << grid_dim.x << "\n"
              << grid_dim.y << "\n"
              << patch_mem << "\n";

    l2_sq<<<grid_dim, block_dim, 2 * patch_mem>>>(d_descriptors, d_centroids, 
                                                  d_l2_squared, n_desc, DESC_DIM,
                                                  n_cent);
    hipDeviceSynchronize();
    cudaCheckError();

    hipMemcpy(&h_l2_squared[0], d_l2_squared, n_desc * n_cent * sizeof(double),
               hipMemcpyDeviceToHost);
    cudaCheckError();

    //hipMemcpy(&h_assignments[0], d_assignments, n_desc * sizeof(int),
    //           hipMemcpyDeviceToHost);

    int i = 0;
    for (auto val : h_l2_squared){ 
        if (i % 32 == 0)
            std::cout << "\n";
        std::cout << val << " ";
        i++;
    }

    hipFree(d_descriptors);
    hipFree(d_centroids);
    hipFree(d_l2_squared);
    hipFree(d_assignments);
    cudaCheckError();

    return h_assignments;
}

}